#include <stdio.h>
#include "gpu_nn.h"

/*
Allocate the memory required by the GPU solver.
Does not handle memory allocation for the data we load in.
*/

void Send_To_Device(float **h_neuron_bias,  float **d_neuron_bias,
                     short **h_neuron_forward_recieve_id, short **d_neuron_forward_recieve_id,
                     short **h_neuron_forward_receive_start, short **d_neuron_forward_receive_start,
                     float ** h_neuron_forward_recieve_weight, float ** d_neuron_forward_recieve_weight,
                     short **h_layer_start, short **d_layer_start,
                     short **h_layer_neuron_type, short **d_layer_neuron_type,
                     float *h_training_data, float **d_training_data, size_t training_data_size,
                     float *h_training_classification, float **d_training_classification, size_t training_class_size,
                     int NO_NEURONS, int NO_WEIGHTS, int NO_LAYERS) {

    // Grab a error type
    hipError_t Error;

    // Send Neuron data to GPU
    size_t size = NO_NEURONS*sizeof(float);
    Error = hipMemcpy(*d_neuron_bias, *h_neuron_bias, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_neuron_bias -> d_neuron_bias) = %s\n", hipGetErrorString(Error));
    
    // Send weight information to GPU
    size = NO_WEIGHTS*sizeof(float);
    Error = hipMemcpy(*d_neuron_forward_recieve_weight, *h_neuron_forward_recieve_weight, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_neuron_forward_recieve_weight -> d_neuron_forward_recieve_weight) = %s\n", hipGetErrorString(Error));    
    
    size = NO_WEIGHTS*sizeof(short);
    Error = hipMemcpy(*d_neuron_forward_recieve_id, *h_neuron_forward_recieve_id, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_neuron_forward_recieve_id -> d_neuron_forward_recieve_id) = %s\n", hipGetErrorString(Error));    

    size = (NO_NEURONS+1)*sizeof(short);
    Error = hipMemcpy(*d_neuron_forward_receive_start, *h_neuron_forward_receive_start, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_neuron_forward_receive_start -> d_neuron_forward_receive_start) = %s\n", hipGetErrorString(Error));

    size = (NO_LAYERS+1)*sizeof(short);
    Error = hipMemcpy(*d_layer_start, *h_layer_start, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_layer_start -> d_layer_start) = %s\n", hipGetErrorString(Error));

    size = NO_LAYERS*sizeof(short);
    Error = hipMemcpy(*d_layer_neuron_type, *h_layer_neuron_type, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_layer_neuron_type -> d_layer_neuron_type) = %s\n", hipGetErrorString(Error));   

    // Training Data
    Error = hipMemcpy(*d_training_data, h_training_data, training_data_size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_training_data -> d_training_data) = %s\n", hipGetErrorString(Error)); 

    Error = hipMemcpy(*d_training_classification, h_training_classification, training_class_size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_training_classification -> d_training_classification) = %s\n", hipGetErrorString(Error)); 

}




void Allocate_Memory(float **h_neuron_bias,  float **d_neuron_bias,
                     float **h_neuron_input, float **d_neuron_input,
                     float **h_neuron_output, float **d_neuron_output,
                     float **h_neuron_delta, float **d_neuron_delta,
                     short **h_neuron_forward_recieve_id, short **d_neuron_forward_recieve_id,
                     short **h_neuron_forward_receive_start, short **d_neuron_forward_receive_start,
                     float ** h_neuron_forward_recieve_weight, float ** d_neuron_forward_recieve_weight,
                     short **h_layer_start, short **d_layer_start,
                     short **h_layer_neuron_type, short **d_layer_neuron_type,
                     float **d_training_data, size_t training_data_size,
                     float **d_training_classification, size_t training_class_size,
                     int NO_NEURONS, int NO_WEIGHTS, int NO_LAYERS) {

    size_t size = NO_NEURONS*sizeof(float);
    *h_neuron_bias = (float*)malloc(size);
    *h_neuron_input = (float*)malloc(size);
    *h_neuron_output = (float*)malloc(size);
    *h_neuron_delta = (float*)malloc(size);

    hipError_t Error;
    Error = hipMalloc((void**)d_neuron_bias, size); 
    printf("CUDA error (malloc d_neuron_bias) = %s\n", hipGetErrorString(Error));
    Error = hipMalloc((void**)d_neuron_input, size); 
    printf("CUDA error (malloc d_neuron_input) = %s\n", hipGetErrorString(Error));
    Error = hipMalloc((void**)d_neuron_output, size); 
    printf("CUDA error (malloc d_neuron_output) = %s\n", hipGetErrorString(Error));
    Error = hipMalloc((void**)d_neuron_delta, size); 
    printf("CUDA error (malloc d_neuron_delta) = %s\n", hipGetErrorString(Error));

    size = NO_WEIGHTS*sizeof(short);
    *h_neuron_forward_recieve_id = (short*)malloc(size);
    Error = hipMalloc((void**)d_neuron_forward_recieve_id, size); 
    printf("CUDA error (malloc d_neuron_forward_recieve_id) = %s\n", hipGetErrorString(Error));

    size = NO_WEIGHTS*sizeof(float);
    *h_neuron_forward_recieve_weight = (float*)malloc(size);
    Error = hipMalloc((void**)d_neuron_forward_recieve_weight, size); 
    printf("CUDA error (malloc h_neuron_forward_recieve_weight) = %s\n", hipGetErrorString(Error));

    size = (NO_NEURONS+1)*sizeof(short);
    *h_neuron_forward_receive_start = (short*)malloc(size);
    Error = hipMalloc((void**)d_neuron_forward_receive_start, size); 
    printf("CUDA error (malloc d_neuron_forward_receive_start) = %s\n", hipGetErrorString(Error));
    
    size = NO_LAYERS*sizeof(short);
    *h_layer_neuron_type = (short*)malloc(size);
    Error = hipMalloc((void**)d_layer_neuron_type, size); 
    printf("CUDA error (malloc d_layer_neuron_type) = %s\n", hipGetErrorString(Error));

    size = (NO_LAYERS+1)*sizeof(short);
    *h_layer_start = (short*)malloc(size);
    Error = hipMalloc((void**)d_layer_start, size); 
    printf("CUDA error (malloc d_layer_start) = %s\n", hipGetErrorString(Error));

    // Detect the size of the training data sets, and allocate that much in Cuda
    Error = hipMalloc((void**)d_training_data, training_data_size); 
    printf("CUDA error (malloc d_training_data) = %s\n", hipGetErrorString(Error));

    Error = hipMalloc((void**)d_training_classification, training_data_size); 
    printf("CUDA error (malloc d_training_classification) = %s\n", hipGetErrorString(Error));

}

void Free_Memory(float **h_neuron_bias,  float **d_neuron_bias,
                     float **h_neuron_input, float **d_neuron_input,
                     float **h_neuron_output, float **d_neuron_output,
                     float **h_neuron_delta, float **d_neuron_delta,
                     short **h_neuron_forward_recieve_id, short **d_neuron_forward_recieve_id,
                     short **h_neuron_forward_receive_start, short **d_neuron_forward_receive_start,
                     float **h_neuron_forward_recieve_weight, float **d_neuron_forward_recieve_weight,
                     short **h_layer_start, short **d_layer_start,
                     short **h_layer_neuron_type, short **d_layer_neuron_type,
                     float **d_training_data, float **d_training_classification) {

    if (*h_neuron_bias) free(*h_neuron_bias);
    if (*h_neuron_input) free(*h_neuron_input);
    if (*h_neuron_output) free(*h_neuron_output);
    if (*h_neuron_delta) free(*h_neuron_delta);
    if (*h_neuron_forward_recieve_id) free(*h_neuron_forward_recieve_id);
    if (*h_neuron_forward_receive_start) free(*h_neuron_forward_receive_start);
    if (*h_neuron_forward_recieve_weight) free(*h_neuron_forward_recieve_weight);
    if (*h_layer_start) free(*h_layer_start);
    if (*h_layer_neuron_type) free(*h_layer_neuron_type);   

    if (*d_neuron_bias) hipFree(*d_neuron_bias);
    if (*d_neuron_input) hipFree(*d_neuron_input);
    if (*d_neuron_output) hipFree(*d_neuron_output);
    if (*d_neuron_delta) hipFree(*d_neuron_delta);
    if (*d_neuron_forward_recieve_id) hipFree(*d_neuron_forward_recieve_id);
    if (*d_neuron_forward_receive_start) hipFree(*d_neuron_forward_receive_start);
    if (*d_neuron_forward_recieve_weight) hipFree(*d_neuron_forward_recieve_weight);

    if (*d_layer_start) hipFree(*d_layer_start);
    if (*d_layer_neuron_type) hipFree(*d_layer_neuron_type);
    if (*d_training_data) hipFree(*d_training_data);
    if (*d_training_classification) hipFree(*d_training_classification);    
}


void Prepare_Network_Size(short *network_layout, short *no_layers, short *no_weights, short *no_neurons) {
    *no_layers = sizeof(network_layout) / sizeof(network_layout[0])-1;
    *no_weights = 0;
    *no_neurons = 0;
    printf("No. of layers = %d\n", *no_layers);
    for (short layer = 0; layer < *no_layers; layer++) {
        printf("Found %d neurons in layer %d\n", network_layout[layer], layer);
        *no_neurons = *no_neurons + network_layout[layer];
    }
    // Compute the number of weights
    for (short layer = 1; layer < *no_layers; layer++) {
        *no_weights += network_layout[layer]*network_layout[layer-1];
    }

    printf("Found a total number of %d neurons\n", *no_neurons);
    printf("Found a total number of %d weights\n", *no_weights);
}

void Prepare_Network_Structure(short *layer_start, short *layer_neuron_type, short *network_layout, 
                               short *neuron_forward_receive_start, short *neuron_forward_recieve_id,
                               float *neuron_bias, float *neuron_forward_recieve_weight,
                               short no_layers, short no_neurons) {

    // The first layer is an input layer (type 0)    
    layer_neuron_type[0] = 0;
    // The last layer is an output layer (type 2)
    layer_neuron_type[no_layers-1] = 2;

    // The remaining types are inner (hidden) layers.
    for (short layer = 1; layer < (no_layers-1); layer++) {
        layer_neuron_type[layer] = 1;
    }

    layer_start[0] = 0;
    // Now to properly compose the layer starts
    for (short layer = 0; layer < no_layers; layer++) {
        layer_start[layer+1] = layer_start[layer] + network_layout[layer];
    }

    // Build up the network now
    // short layer_start[NO_LAYERS+1] = {0, 4, 9, 12};
    // short layer_neuron_type[NO_LAYERS] = {0, 1, 2}; // 0 = input, 1 = hidden, 2 = output
    // short neuron_forward_recieve_id[] =    {0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3, 0, 1, 2, 3, 4, 5, 6, 7, 8, 4, 5, 6, 7, 8, 4, 5, 6, 7, 8};
    // short neuron_forward_receive_start[] = {0, 0, 0, 0, 0, 4, 8, 12, 16, 20, 25, 30, 35};
    neuron_forward_receive_start[0] = 0;
    short cumulative_sum_neurons = 0;
    short cumulative_input_sum = 0;
    short weight_count = 0;
    for (short layer = 0; layer <= no_layers; layer++) {
        // And we iterate over the neurons in this layer
        for (short layer_neuron = 0; layer_neuron < (layer_start[layer+1] - layer_start[layer]); layer_neuron++) {
            short neuron_id = layer_start[layer] + layer_neuron;
            if (layer_neuron_type[layer] == 0) {
                // This is an input layer
                neuron_forward_receive_start[neuron_id+1] = 0; // There are no inputs
            } else {
                short no_inputs = network_layout[layer-1];
                for (short input = 0; input < no_inputs; input++) {
                    short source_id = cumulative_sum_neurons + input;
                    neuron_forward_recieve_id[weight_count] = source_id;
                    weight_count++;
                }
                cumulative_input_sum += no_inputs;
                neuron_forward_receive_start[neuron_id+1] = cumulative_input_sum;
            }
        }
        if (layer_neuron_type[layer] > 0) {
            cumulative_sum_neurons += layer_start[layer]; // Keeps track of the id of the last neuron in the previous layer
        }
    }

    // Initialise the bias and weights values
    srand(time(NULL));
    for (int i = 0; i < no_neurons; i++) {
        neuron_bias[i] = (float)rand()/RAND_MAX;
    }
    for (int i = 0; i < weight_count; i++) {
        neuron_forward_recieve_weight[i] = (float)rand()/RAND_MAX;
    }

}  





void Train_Network(float *h_training_data, size_t training_data_size, float *h_training_classification, size_t training_class_size, short *network, int NO_SAMPLES, int NO_EPOCHS, float learning_rate) {

    float *h_neuron_bias, *d_neuron_bias;
    float *h_neuron_input, *d_neuron_input;
    float *h_neuron_output, *d_neuron_output;
    float *h_neuron_delta, *d_neuron_delta;
    short *h_neuron_forward_recieve_id, *d_neuron_forward_recieve_id;
    short *h_neuron_forward_receive_start, *d_neuron_forward_receive_start;
    float *h_neuron_forward_recieve_weight, *d_neuron_forward_recieve_weight;
    short *h_layer_neuron_type, *d_layer_neuron_type;
    short *h_layer_start, *d_layer_start;
    short no_weights, no_layers, no_neurons;

    // Prepare memory for the training data and classification
    float *d_training_data, *d_training_classification;

    Prepare_Network_Size(network, &no_layers, &no_weights, &no_neurons);

    printf("Idenfified %d layers, %d neurons and %d weights\n", no_layers, no_neurons, no_weights);

    Allocate_Memory(&h_neuron_bias,  &d_neuron_bias,
                     &h_neuron_input, &d_neuron_input,
                     &h_neuron_output, &d_neuron_output,
                     &h_neuron_delta, &d_neuron_delta,
                     &h_neuron_forward_recieve_id, &d_neuron_forward_recieve_id,
                     &h_neuron_forward_receive_start, &d_neuron_forward_receive_start,
                     &h_neuron_forward_recieve_weight, &d_neuron_forward_recieve_weight,
                     &h_layer_start, &d_layer_start,
                     &h_layer_neuron_type, &d_layer_neuron_type,
                     &d_training_data, training_data_size,
                     &d_training_classification, training_class_size,
                     no_neurons, no_weights, no_layers);

    Prepare_Network_Structure(h_layer_start, h_layer_neuron_type, network,
                             h_neuron_forward_receive_start, h_neuron_forward_recieve_id, 
                             h_neuron_bias, h_neuron_forward_recieve_weight,
                             no_layers, no_neurons);
    printf("Training network\n");


    // Send things to the GPU
    Send_To_Device(&h_neuron_bias,  &d_neuron_bias,
                   &h_neuron_forward_recieve_id, &d_neuron_forward_recieve_id,
                   &h_neuron_forward_receive_start, &d_neuron_forward_receive_start,
                   &h_neuron_forward_recieve_weight, &d_neuron_forward_recieve_weight,
                   &h_layer_start, &d_layer_start,
                   &h_layer_neuron_type, &d_layer_neuron_type,
                   h_training_data, &d_training_data, training_data_size,
                   h_training_classification, &d_training_classification, training_class_size,
                   no_neurons, no_weights, no_layers);

    // 

    for (int epoch = 0; epoch < NO_EPOCHS; epoch++) {
        // Employ Data Parallelism
        // Each thread will train (using forward and backward propagation) one sample

        int threads_per_block = 32;
        int no_blocks = NO_SAMPLES/threads_per_block;


    }




    Free_Memory(&h_neuron_bias,  &d_neuron_bias,
        &h_neuron_input, &d_neuron_input,
        &h_neuron_output, &d_neuron_output,
        &h_neuron_delta, &d_neuron_delta,
        &h_neuron_forward_recieve_id, &d_neuron_forward_recieve_id,
        &h_neuron_forward_receive_start, &d_neuron_forward_receive_start,
        &h_neuron_forward_recieve_weight, &d_neuron_forward_recieve_weight,
        &h_layer_start, &d_layer_start,
        &h_layer_neuron_type, &d_layer_neuron_type,
        &d_training_data, &d_training_classification);

}